#include "hip/hip_runtime.h"
#include <>
#include <iostream>

#include "cu_func.cuh"

// 两个向量加法kernel，grid和block均为一维
__global__ void add_kernel(float* x, float* y, float* z, int N)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) {
        z[i] = x[i] + y[i];
    }
}

// x: [M, P]
// y: [P, N]
// ret z: [M, N]
__global__ void multi_kernel(float* x, float* y, float* z, int M, int N, int P)
{
    int i1 = blockIdx.x;
    int i2 = threadIdx.x;
    int s1 = (M + gridDim.x - 1) / gridDim.x;
    int s2 = (N + blockDim.x - 1) / blockDim.x;

    for (int i = i1; i < M; i += s1) {
        for (int j = i2; j < N; j += s2) {
            float sum = 0.0f;
            for (int m = 0; m < P; m++) {
                sum += x[i * P + m] * y[m * N + j];
            }
            z[i * M + j] = sum;
        }
    }
}

void vectorAdd(float* x, float* y, float* z, int n)
{
    float* d_x;
    float* d_y;
    float* d_z;
    size_t nBytes = n * sizeof(float);
    hipMalloc(&d_x, nBytes);
    hipMalloc(&d_y, nBytes);
    hipMalloc(&d_z, nBytes);

    hipMemcpy(d_x, x, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, nBytes, hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / (blockSize.x));
    add_kernel<<<gridSize, blockSize>>>(d_x, d_y, d_z, n);

    hipMemcpy(z, d_z, nBytes, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
}

// x: [M, P]
// y: [P, N]
// ret z: [M, N]
void matrixMulti(float* x, float* y, float* z, int M, int N, int P)
{
    float* d_x;
    float* d_y;
    float* d_z;

    size_t byte_x = M * P * sizeof(float);
    size_t byte_y = P * N * sizeof(float);
    size_t byte_z = M * N * sizeof(float);

    hipMalloc(&d_x, byte_x);
    hipMalloc(&d_y, byte_y);
    hipMalloc(&d_z, byte_z);

    hipMemcpy(d_x, x, byte_x, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, byte_y, hipMemcpyHostToDevice);

    dim3 blockSize(32);
    dim3 gridSize(32);
    multi_kernel<<<gridSize, blockSize>>>(d_x, d_y, d_z, M, N, P);

    hipMemcpy(z, d_z, byte_z, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
}


int main1()
{
    constexpr int N = 1 << 20;
    int nBytes      = N * sizeof(float);
    // 申请host内存
    float *x, *y, *z;
    x = (float*)malloc(nBytes);
    y = (float*)malloc(nBytes);
    z = (float*)malloc(nBytes);

    // 初始化数据
    for (int i = 0; i < N; ++i) {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    // 申请device内存
    float *d_x, *d_y, *d_z;
    hipMalloc((void**)&d_x, nBytes);
    hipMalloc((void**)&d_y, nBytes);
    hipMalloc((void**)&d_z, nBytes);

    // 将host数据拷贝到device
    hipMemcpy((void*)d_x, (void*)x, nBytes, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_y, (void*)y, nBytes, hipMemcpyHostToDevice);
    // 定义kernel的执行配置
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    // 执行kernel
    add_kernel<<<gridSize, blockSize>>>(d_x, d_y, d_z, N);

    // 将device得到的结果拷贝到host
    hipMemcpy((void*)z, (void*)d_z, nBytes, hipMemcpyDeviceToHost);

    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(z[i] - 30.0));
    std::cout << "最大误差: " << maxError << std::endl;

    // 释放device内存
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    // 释放host内存
    free(x);
    free(y);
    free(z);

    return 0;
}

int main2()
{
    int N      = 1 << 20;
    int nBytes = N * sizeof(float);

    // 申请托管内存
    float *x, *y, *z;
    hipMallocManaged((void**)&x, nBytes);
    hipMallocManaged((void**)&y, nBytes);
    hipMallocManaged((void**)&z, nBytes);

    // 初始化数据
    for (int i = 0; i < N; ++i) {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    // 定义kernel的执行配置
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    // 执行kernel
    add_kernel<<<gridSize, blockSize>>>(x, y, z, N);

    // 同步device 保证结果能正确访问
    hipDeviceSynchronize();
    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(z[i] - 30.0));
    std::cout << "最大误差: " << maxError << std::endl;

    // 释放内存
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}
